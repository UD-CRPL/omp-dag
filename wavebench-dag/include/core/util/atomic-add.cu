#include "hip/hip_runtime.h"
#include "atomic-add.h"
#ifdef CUDA_SUPPORT_COREQ
#if __CUDA_ARCH__ < 600
__device__ double __core__::atomicAdd(double* address, double val) {
	unsigned long long int* address_as_ull=(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull,assumed;
	do{
		assumed=old;
		old=atomicCAS(address_as_ull,assumed,__double_as_longlong(val+__longlong_as_double(assumed)));
	} while(assumed!=old);
	return __longlong_as_double(old);
}
#endif
#endif
